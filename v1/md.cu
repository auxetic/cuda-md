#include "hip/hip_runtime.h"
#include "md.h"

// internal function
__global__ void kernel_calc_chi( tpvec *tdconv, tpvec *tdconf, int natom );
hipError_t gpu_calc_chi( tpvec *tdconv, tpvec *tdconf, tpbox tbox, double *chi );
__global__ void kernel_modify_force( tpvec *tdconf, tpvec *tdconv, int natom, double tchi );
hipError_t gpu_modify_force( tpvec *tdconf, tpvec *tdconv, tpbox tbox, double tchi );

// internal variables
#define dt 0.01
tpvec  *dconv, *dconf;

// kernel varialbes
#define BLOCK_SIZE_256  256 
__managed__ double mpp, mpf;

tpmdset mdset;


void init_nvt( tpvec *thcon, double *thradius, tpbox tbox, double ttemper )
    {
    // allocate config array
    hipMalloc((void **)&dcon    , sizeof(tpvec)  * tbox.natom );
    hipMalloc((void **)&dconv   , sizeof(tpvec)  * tbox.natom );
    hipMalloc((void **)&dconf   , sizeof(tpvec)  * tbox.natom );
    hipMalloc((void **)&dradius , sizeof(double) * tbox.natom );

    hipMemcpy(dcon,    thcon,    sizeof(tpvec)*tbox.natom,  hipMemcpyHostToDevice);
    hipMemcpy(dradius, thradius, sizeof(double)*tbox.natom, hipMemcpyHostToDevice);

    mdset.temper = ttemper;
    }
    
void gpu_run_nvt( tpbox tbox, double ttemper, int steps )
    {
    for ( int step=1; step <= steps; step++ )
        {
        // check and make list
        if ( gpu_check_list( dcon, tbox, dlist ) )
            {
            printf( "making list \n" );
            gpu_make_hypercon( dcon, dradius, tbox, dblocks, hblockset );
            gpu_make_list( dlist, dblocks, dcon, hblockset, tbox );
            }

        // velocity verlet / integrate veclocity and config
        gpu_update_vr( dcon, dconv, dconf, tbox, dt );

        // temp
        double press;
        // calc force
        gpu_calc_force( dlist, dcon, dradius, dconf, &press, tbox );

        // nvt / modify force
        double chi;
        gpu_calc_chi( dconv, dconf, tbox, &chi );
        gpu_modify_force( dconf, dconv, tbox, chi );

        // velocity verlet / integrate velocity
        gpu_update_v( dconv, dconf, tbox, dt );
        }
    }

hipError_t gpu_calc_chi( tpvec *tdconv, tpvec *tdconf, tpbox tbox, double *chi )
    {
    const int block_size = 256;
    const int natom = tbox.natom;

    dim3 grids( ceil( natom / block_size )+1, 1, 1 );
    dim3 threads( block_size, 1, 1 );

    kernel_calc_chi <<< grids, threads >>> ( tdconv, tdconf, natom );

    hipError_t err;
    err = hipDeviceSynchronize();

    if ( err != hipSuccess )
        {
        fprintf(stderr, "cudaDeviceSync failed, %s, %d, err = %d\n", __FILE__, __LINE__, err);
        }
    
    *chi = mpf/mpp;

    return err;
    }

__global__ void kernel_calc_chi( tpvec *tdconv, tpvec *tdconf, int natom )
    {
    __shared__ double spp[BLOCK_SIZE_256];
    __shared__ double spf[BLOCK_SIZE_256];

    const int tid = threadIdx.x;
    const int i   = tid + blockIdx.x * blockDim.x;

    spp[tid] = 0.0;
    spf[tid] = 0.0;

    if ( i < natom )
        {
        double fxi = tdconf[i].x;
        double fyi = tdconf[i].y;
        double vxi = tdconv[i].x;
        double vyi = tdconv[i].y;
        spp[i] = vxi * vxi + vyi * vyi;
        spf[i] = vxi * fxi + vyi * fyi;
        }

    int j = blockDim.x;
    j >>= 1;
    while ( j != 0 )
        {
        if ( tid < j )
            {
            spp[tid] += spp[tid+j];
            spf[tid] += spf[tid+j];
            }
        __syncthreads();
        j >>= 1;
        }

    if ( tid == 0 )
        {
        atomicAdd( &mpp, spp[0] );
        atomicAdd( &mpf, spf[0] );
        }

    }

hipError_t gpu_modify_force( tpvec *tdconf, tpvec *tdconv, tpbox tbox, double tchi )
    {
    const int block_size = 256;
    const int natom = tbox.natom;

    dim3 grids( ceil(natom/block_size)+1, 1, 1 );
    dim3 threads( block_size, 1, 1 );
    
    kernel_modify_force <<< grids, threads >>> ( tdconf, tdconv, natom, tchi );

    hipError_t err;
    err = hipDeviceSynchronize();

    if ( err != hipSuccess )
        {
        fprintf(stderr, "cudaDeviceSync failed, %s, %d, err = %d\n", __FILE__, __LINE__, err);
        }
    
    return err;
    }

__global__ void kernel_modify_force( tpvec *tdconf, tpvec *tdconv, int natom, double tchi )
    {
    const int tid = threadIdx.x;
    const int i   = tid + blockIdx.x * blockDim.x;

    if ( i < natom )
        {
        tdconf[i].x = tdconf[i].x - tchi * tdconv[i].x;
        tdconf[i].y = tdconf[i].y - tchi * tdconv[i].y;
        }

    }

