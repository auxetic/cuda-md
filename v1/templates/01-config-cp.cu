#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "system.h"
#include "config.h"
#include "list.h"
#include "fire.h"

int main(void)
    {

    int deviceno, natom, seed;
    double press;
    char foutput[100];

    scanf("%d", &deviceno);
    scanf("%d", &natom );
    scanf("%d", &seed );
    scanf("%le", &press );
    scanf("%s", foutput );

    hipSetDevice(deviceno);
    // set
    box.natom = natom;
    box.phi = 0.86;
    sets.seed = seed;

    printf("%d\n", deviceno);
    printf("%d\n", natom);
    printf("%d\n", seed);
    printf("%e\n", press);
    printf("%s\n", foutput);

//exit(0);
    // cpu config
    //printf( "allocate *con, generate a random config\n" );
    alloc_con( &con, &radius, box.natom );
    gen_config( con, radius, &box, sets );

    // fire
    mini_fire_cp( con, radius, &box, press );

    FILE *fio;
    fio = fopen(foutput, "w+");
        fprintf( fio, "%d %26.16e \n", box.natom, box.x );
        for ( int i=0; i<box.natom; i++ )
            fprintf( fio, "%26.16e  %26.16e  %26.16e \n", con[i].x*box.xinv, con[i].y*box.yinv, radius[i]*box.xinv );
    fclose(fio);

    return 0;
    }
    //// fire on gpu
    //mini_fire_cv( con, radius, box );
