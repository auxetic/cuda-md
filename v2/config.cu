#include "hip/hip_runtime.h"
#include "config.h"

// variables define
// host
tpvec  *con;
double *radius;
// device
tpvec  *dcon;
double *dradius;


// allocate memory space of config
void alloc_con( tpvec **tcon, double **tradius, int tnatom )
    {
    *tcon    = (tpvec *)malloc( tnatom*sizeof(tpvec)  );
    *tradius = (double*)malloc( tnatom*sizeof(double) );
    }

// allocate memory space of config
hipError_t device_alloc_con( tpvec **tcon, double **tradius, int tnatom )
    {
    check_cuda( hipMallocManaged( &tcon,     tnatom*sizeof(tpvec)  ) );
    check_cuda( hipMallocManaged( &tradius , tnatom*sizeof(double) ) );
    return hipSuccess;
    }

// generate random config on host
void gen_config( tpvec *tcon, double *tradius, tpbox *tbox, tpsets tsets )
    {
    // 1. intiate random number generator;
    srand(tsets.seed);

    // 2. set atom radius
    for ( int i=0; i<tbox->natom; i++ )
        {
        if ( i < tbox->natom/2 )
            tradius[i] = 0.5;
        else
            tradius[i] = 0.5 * ratio;
        }

    // 3. calc area of disks
    double sdisk = 0.0;
    for ( int i=0; i<tbox->natom; i++ )
        {
        sdisk += tradius[i]*tradius[i];
        }
    sdisk *= Pi;

    // 4. cal box.l from phi
    double vol     = sdisk / tbox->phi;
    double lx      = sqrt(vol);
    tbox->len.x    = lx;
    tbox->len.y    = lx;
    tbox->leninv.x = 1.0 / lx;
    tbox->leninv.y = 1.0 / lx;
    tbox->strain   = 0.0;

    // 5. give a random config
    for ( int i=0; i<tbox->natom; i++ )
        {
        tcon[i].x = ( (double)rand()/RAND_MAX - 0.5 ) * lx;
        tcon[i].y = ( (double)rand()/RAND_MAX - 0.5 ) * lx;
        }
    }

// read config
void read_config( FILE *tfio, tpvec *tcon, double *tradius, tpbox *tbox )
    {
    int natom;
    fscanf(tfio, "%d", &natom);
    tbox->natom = natom;
    double lx;
    fscanf(tfio, "%le", &lx);
    tbox->len.x = lx;
    tbox->len.y = lx;
    tbox->leninv.x = 1e0/lx;
    tbox->leninv.y = 1e0/lx;
    tbox->strain = 0e0;

    for ( int i=0; i<natom; i++ )
        {
        double x, y, r;
        fscanf(tfio, "%le", &x);
        fscanf(tfio, "%le", &y);
        fscanf(tfio, "%le", &r);
        tcon[i].x  = x*lx;
        tcon[i].y  = y*lx;
        tradius[i] = r*lx;
        }
    }

// move all atoms to central box
void trim_config( tpvec *tcon, tpbox tbox )
    {
    double lx = tbox.len.x;
    double ly = tbox.len.y;
    for ( int i=0; i<tbox.natom; i++ )
        {
        double cory;
        cory = round( tcon[i].y / ly );
        tcon[i].x -= cory * ly * tbox.strain;

        tcon[i].x -= round( tcon[i].x / lx ) * lx;
        tcon[i].y -= cory * ly;
        }
    }

__global__ void kernel_trim_config( tpvec *tcon, int tnatom, double lx, double ly )
    {
    const int i   = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i < tnatom )
        {
        double x, y;
        x = tcon[i].x;
        y = tcon[i].y;

        x -= round( x/lx ) * lx;
        y -= round( y/ly ) * ly;

        tcon[i].x = x;
        tcon[i].y = y;
        }
    }

hipError_t gpu_trim_config( tpvec *tcon, tpbox tbox )
    {
    const int    natom = tbox.natom;
    const double lx    = tbox.len.x;
    const double ly    = tbox.len.y;

    const int    block_size = 256;
    dim3 grids( (natom/block_size)+1, 1, 1 );
    dim3 threads( block_size, 1, 1 );

    kernel_trim_config<<< grids, threads >>>( tcon, natom, lx, ly );
    check_cuda( hipDeviceSynchronize() );

    return hipSuccess;
    }

//// copy config from host to device
    //hipError_t trans_con_to_gpu( tpvec *tdcon, double *tdradius, int tnatom,
    //                              tpvec *thcon, double *thradius )
    //    {
    //    hipError_t err;
    //    err = hipMemcpy(tdcon, thcon, tnatom*sizeof(tpvec), hipMemcpyHostToDevice);
    //    if ( err != hipSuccess )
    //        {
    //        fprintf(stderr,"hipMemcpy failed in %s, %d, err=%d\n", __FILE__, __LINE__, err);
    //        exit(-1);
    //        }
    //    err = hipMemcpy(tdradius, thradius, tnatom*sizeof(double), hipMemcpyHostToDevice);
    //    if ( err != hipSuccess )
    //        {
    //        fprintf(stderr,"hipMemcpy failed in %s, %d, err=%d\n", __FILE__, __LINE__, err);
    //        exit(-1);
    //        }
    //    return hipSuccess;
    //    }

//// copy config from device to host
    //hipError_t trans_con_to_host( tpvec *thcon, double *thradius, int tnatom,
    //                               tpvec *tdcon, double *tdradius )
    //    {
    //    hipError_t err;
    //    err = hipMemcpy(thcon, tdcon, tnatom*sizeof(tpvec), hipMemcpyDeviceToHost);
    //    if ( err != hipSuccess )
    //        {
    //        fprintf(stderr,"hipMemcpy failed in %s, %d\n", __FILE__, __LINE__);
    //        exit(-1);
    //        }
    //    err = hipMemcpy(thradius, tdradius, tnatom*sizeof(double), hipMemcpyDeviceToHost);
    //    if ( err != hipSuccess )
    //        {
    //        fprintf(stderr,"hipMemcpy failed in %s, %d\n", __FILE__, __LINE__);
    //        exit(-1);
    //        }
    //    return hipSuccess;
    //    }
