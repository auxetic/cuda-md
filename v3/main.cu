#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "system.h"
#include "config.h"
#include "mdfunc.h"

int main(void)
    {
    box_t  box;
    box.natom = 16384;
    box.phi   = 0.88;
    sets_t sets;
    sets.seed = 1;
    hipSetDevice(0);

    double  press = 0.0;
    double  *radius = NULL;
    vec_t   *con    = NULL;
    vec_t   *conf   = NULL;
    check_cuda( hipMallocManaged( &con,     box.natom*sizeof(vec_t)  ) );
    check_cuda( hipMallocManaged( &conf,    box.natom*sizeof(vec_t)  ) );
    check_cuda( hipMallocManaged( &radius , box.natom*sizeof(double) ) );
    gen_config( con, radius, &box, sets );

    hycon_t hycon;
    calc_hypercon_args( &hycon, box );
    check_cuda( hipMallocManaged( &hycon.blocks, hycon.args.nblocks*sizeof(block_t) ) );
    printf("h\n");//debug
    gpu_make_hypercon( hycon, con, radius, box);
    map( hycon );


    FILE *fptr= fopen("i_con.dat", "w+");
    write_config( fptr, con, radius, &box );
    fclose(fptr);


    gpu_calc_force( conf, &hycon, &press, box );
    printf("%26.16e\n", press);

    fptr= fopen("i_conf.dat", "w+");
    write_config( fptr, conf, radius, &box );
    fclose(fptr);

    return 0;
    }
