#include "hip/hip_runtime.h"
#include "mdfunc.h"

__managed__ double g_fmax;
__managed__ double g_wili;

<<<<<<< HEAD


// calculate force of one block with all its neighbour at once
__global__ void kernel_calc_force_all_neighb_block( vec_t        *conf, 
                                                    block_t      *blocks, 
                                                    const int    tbid, 
                                                    const double tlx )
    {
    __shared__ double sm_wili;

    __shared__ block_t block_core;
    __shared__ block_t block_edge;

    const int i   = threadIdx.x + blockIdx.x * blockDim.x;
    const int tid = threadIdx.x;

    if ( tid == 0 ) sm_wili = 0.0;

    rxi[tid] = tdblocks[tbid].rx[i];
    ryi[tid] = tdblocks[tbid].ry[i];
    rzi[tid] = tdblocks[tbid].rz[i];
    ri [tid] = tdblocks[tbid].radius[i];
    double fx = 0.0;
    double fy = 0.0;
    double fz = 0.0;
    double wi = 0.0;

    __syncthreads();

    int j;
    double rxij, ryij, rzij, rij, dij, Vr;
<<<<<<< HEAD
    //block_t *blocki, *blockj;
    blocki = tdblocks[tbid];
=======
    cell_t *blocki, *blockj;
    blocki = &tdblocks[tbid];
>>>>>>> debug
    // self block force
    rxj[tid] = rxi[tid];
    ryj[tid] = ryi[tid];
    rzj[tid] = rzi[tid];
    for ( int j=0; j<tdblocks[tbid].natom; j++)
        {
        rxij  =rxj[j]-rxi[tid];
        ryij  =ryj[j]-ryi[tid];
        rzij  =rzj[j]-rzi[tid];
        rxij -=round(rxij/tlx)*tlx;
        ryij -=round(ryij/tlx)*tlx;
        rzij -=round(rzij/tlx)*tlx;

        rij = rxij*rxij + ryij*ryij + rzij*rzij;
        dij = ri[tid] + rj[j];
        
        if ( tid < blocki->natom && tid != j )
            {
            if ( rij < dij*dij )
                {
                rij = sqrt(rij);

                Vr = - ( 1.0 - rij/dij ) / dij;

                fx -= - Vr * rxij / rij;
                fy -= - Vr * ryij / rij;
                fz -= - Vr * rzij / rij;

                // wili
                wi += - Vr * rij;
                }
            }

        }
    // joint block force
    int bidj;
    for ( int jj=0; jj<26; jj++ )
        {
        bidj = tdblocks[tbid].neighb[jj];
        rxj[tid] = tdblocks[bidj].rx[tid];
        ryj[tid] = tdblocks[bidj].ry[tid];
        rzj[tid] = tdblocks[bidj].rz[tid];
        rj[tid]  = tdblocks[bidj].radius[tid];

        for ( int j = 0; j < tdblocks[bidj].natom; j++)
            {
            rxij  =rxj[j]-rxi[tid];
            ryij  =ryj[j]-ryi[tid];
            rzij  =rzj[j]-rzi[tid];
            rxij -=round(rxij/tlx)*tlx;
            ryij -=round(ryij/tlx)*tlx;
            rzij -=round(rzij/tlx)*tlx;

            rij = rxij*rxij + ryij*ryij + rzij*rzij;
            dij = ri[tid] + rj[j];

            if ( tid < tdblocks[tbid].natom )
                {
                if ( rij < dij*dij )
                    {
                    rij = sqrt(rij);

                    Vr = - ( 1.0 - rij/dij ) / dij;

                    fx -= - Vr * rxij / rij;
                    fy -= - Vr * ryij / rij;
                    fz -= - Vr * rzij / rij;

                    // wili
                    wi += - Vr * rij;
                    }
                }
            }
        }

    conf[i].x = fx;
    conf[i].y = fy;
    conf[i].z = fz;

    atomicAdd( &sm_wili, wi );

    __syncthreads();
    if ( threadIdx.x == 0 )
        {
        //sm_wili /= (double ) sysdim;
        atomicAdd( &g_wili, sm_wili );
        }

    }


hipError_t gpu_calc_force( vec_t   *conf, 
                            hycon_t *hycon, 
                            double  *static_press, 
                            box_t   box )
    {
    const int block_size = 128;

    const int natom = box.natom;
    const double lx = box.len.x;

    const int nblocks = thdblocks->args.nblocks;
    const int nblockx = thdblocks->args.nblock.x;

    check_cuda( hipDeviceSynchronize() );
    g_wili = 0.0;


    block_t *block;
    int grids, threads;
    for ( int i = 0; i < nblocks; i++ )
        {
        grids   = (nblocks/block_size)+1;
        threads = block_size;
<<<<<<< HEAD
        kernel_calc_force_all_neighb_block <<<grids, threads >>> ( conf, thdblocks.oneblocks, i, lx);
=======
        kernel_calc_force_all_neighb_block <<<grids, threads >>> ( thdconf, thdblocks->oneblocks, i, lx);
>>>>>>> debug
        }

    check_cuda( hipDeviceSynchronize() );

    *static_press = g_wili / (double) sysdim / pow(lx, sysdim);

    return hipSuccess;
    }


<<<<<<< HEAD
__global__ void kernel_calc_fmax( vec_t *conf, int natom )
    {
    __shared__ double block_f[256];
    const int tid = threadIdx.x;
    const int i   = threadIdx.x + blockIdx.x * blockDim.x;

    block_f[tid] = 0.0;

    if ( i < natom )
        block_f[tid] = fmax( fabs(conf[i].x), fabs(conf[i].y) );

    __syncthreads();

    int j = 256;
    j >>= 1;
    while ( j != 0 )
        {
        if ( tid < j )
            {
            block_f[tid] = fmax( block_f[tid], block_f[tid+j] );
            }
        __syncthreads();
        j >>= 1;
        }

    if ( tid == 0 )
        atomicMax( &g_fmax, block_f[0] );
    }

double gpu_calc_fmax( vec_t *conf, box_t box )
    {
    const int block_size = 256;
    const int natom = box.natom;

    g_fmax = 0.0;

    dim3 grids( (natom/block_size)+1, 1, 1);
    dim3 threads( block_size, 1, 1);
    kernel_calc_fmax <<< grids, threads >>> ( conf, natom );
    check_cuda( hipDeviceSynchronize() );
=======

>>>>>>> debug

